#include "hip/hip_runtime.h"
/*    Original license:
 *
 *    Copyright 2021 ASTRON
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 */

/* This code is based on
 * https://git.astron.nl/RD/tensor-core-correlator/-/blob/83abdcc/libtcc/TCCorrelator.cu
 *
 * See https://developer.nvidia.com/gtc/2019/video/s9306 for a high-level overview.
 * Lower-level details are in the doc/xbgpu.tcc.rst (and built by Sphinx with
 * the rest of the documentation).
 *
 * It has been modified by SARAO:
 * - Wrap the file in extern "C++" to make it work with PyCUDA (see below)
 * - Add results to the output instead of overwriting, to allow accumulation
 *   across multiple calls (with saturation rather than wrapping).
 * - Conjugate the output, to provide the other triangle of the visibility
 *   matrix.
 * - Take the input axes in a different order.
 * - Remove trailing whitespace.
 *
 * SARAO's modification is licenced as follows:
 *******************************************************************************
 * Copyright (c) 2020-2021, National Research Foundation (SARAO)
 *
 * Licensed under the BSD 3-Clause License (the "License"); you may not use
 * this file except in compliance with the License. You may obtain a copy
 * of the License at
 *
 *   https://opensource.org/licenses/BSD-3-Clause
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ******************************************************************************/

/* PyCUDA wraps the whole file in 'extern "C"', but most of the code expects
 * C++ linkage. So we wrap the whole original file in 'extern "C++"' to cancel
 * that out.
 *
 * When this code gets closer to production, the suggested fix is to modify the
 * accel.build() and context.compile() functions in katsdpsigproc to take a
 * no_extern_c flag as these ones are the methods that will call the
 * pycuda.compiler.SourceModule(...) constructor.
 */
extern "C++" {

#if 1000 * __CUDACC_VER_MAJOR__ + __CUDACC_VER_MINOR__ >= 11001 && __CUDA_ARCH__ >= 800
#define ASYNC_COPIES
#endif

#if defined ASYNC_COPIES
#include <cooperative_groups/memcpy_async.h>
#endif

#include <mma.h>

#define NR_BASELINES		(NR_RECEIVERS * (NR_RECEIVERS + 1) / 2)
#define ALIGN(A,N)		(((A)+(N)-1)/(N)*(N))

#define NR_TIMES_PER_BLOCK	(128 / (NR_BITS))
#define NR_RECEIVERS_PER_TCM_X	((NR_BITS) == 4 ? 2 : 4)
#define NR_RECEIVERS_PER_TCM_Y	((NR_BITS) == 4 ? 4 : 8)

#define COMPLEX			2

#if __CUDA_ARCH__ < (NR_BITS == 4 ? 730 : NR_BITS == 8 ? 720 : NR_BITS == 16 ? 700 : 0)
#error this architecture has no suitable tensor cores
#endif

#if __CUDA_ARCH__ != 700 && __CUDA_ARCH__ != 720 && __CUDA_ARCH__ != 750 && __CUDA_ARCH__ != 800 && __CUDA_ARCH__ != 860
#define PORTABLE // unknown architecture -> write visibilities in portable way (via shared memory)
#endif

#if NR_RECEIVERS_PER_BLOCK != 32 && NR_RECEIVERS_PER_BLOCK != 48 && NR_RECEIVERS_PER_BLOCK != 64
#error unsupported NR_RECEIVERS_PER_BLOCK
#endif

#if NR_SAMPLES_PER_CHANNEL % NR_TIMES_PER_BLOCK != 0
#error NR_SAMPLES_PER_CHANNEL should be a multiple of NR_TIMES_PER_BLOCK
#endif

#define MIN(A,B) ((A)<(B)?(A):(B))


using namespace nvcuda::wmma;

#if NR_BITS == 4
typedef char    Sample;
typedef int2    Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];
#elif NR_BITS == 8
typedef char2   Sample;
typedef int2    Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];
#elif NR_BITS == 16
typedef __half2 Sample;
typedef float2  Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];
#endif
typedef Sample Samples[NR_RECEIVERS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_TIMES_PER_BLOCK][NR_POLARIZATIONS];

#if NR_BITS == 4
typedef fragment<matrix_a, 8, 8, 32, experimental::precision::s4, row_major> Afrag;
typedef fragment<matrix_b, 8, 8, 32, experimental::precision::s4, col_major> Bfrag;
typedef fragment<accumulator, 8, 8, 32, int>                                 Sum;
#elif NR_BITS == 8
typedef fragment<matrix_a, 16, 16, 16, signed char, row_major>               Afrag;
typedef fragment<matrix_b, 16, 16, 16, signed char, col_major>               Bfrag;
typedef fragment<accumulator, 16, 16, 16, int>                               Sum;
#elif NR_BITS == 16
typedef fragment<matrix_a, 16, 16, 16, __half, row_major>                    Afrag;
typedef fragment<matrix_b, 16, 16, 16, __half, col_major>                    Bfrag;
typedef fragment<accumulator, 16, 16, 16, float>                             Sum;
#endif


#if NR_BITS == 4
typedef int2   ScratchSpace[4][NR_POLARIZATIONS][2][NR_POLARIZATIONS];
#elif NR_BITS == 8
typedef int2   ScratchSpace[8][NR_POLARIZATIONS][4][NR_POLARIZATIONS];
#elif NR_BITS == 16
typedef float2 ScratchSpace[8][NR_POLARIZATIONS][4][NR_POLARIZATIONS];
#endif


__device__ inline int conj_perm(int v)
{
#if NR_BITS == 4
  //return ((v & 0x0F0F0F0F) << 4) | (__vnegss4(v >> 4) & 0x0F0F0F0F);
  return ((v & 0x0F0F0F0F) << 4) | ((0xF0F0F0F0 - ((v >> 4) & 0x0F0F0F0F)) & 0x0F0F0F0F);
#elif NR_BITS == 8
  //return __byte_perm(v, __vnegss4(v), 0x2705);
  return __byte_perm(v, 0x00FF00FF - (v & 0xFF00FF00), 0x2705);
#elif NR_BITS == 16
  return __byte_perm(v ^ 0x80000000, v, 0x1032);
#endif
}


__device__ inline int2 conj_perm(int2 v)
{
  return make_int2(conj_perm(v.x), conj_perm(v.y));
}


__device__ inline int4 conj_perm(int4 v)
{
  return make_int4(conj_perm(v.x), conj_perm(v.y), conj_perm(v.z), conj_perm(v.w));
}


#if defined ASYNC_COPIES
#define READ_AHEAD        MIN(2, NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK)
#define NR_SHARED_BUFFERS MIN(4, NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK)
#else
#define READ_AHEAD        1
#define NR_SHARED_BUFFERS 2
#endif


template <unsigned nrReceiversPerBlock = NR_RECEIVERS_PER_BLOCK> struct SharedData
{
#if NR_BITS == 4
  typedef char        Asamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK][1];
  typedef char        Bsamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][COMPLEX][NR_TIMES_PER_BLOCK + 16][1];
#elif NR_BITS == 8
  typedef signed char Asamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK][COMPLEX];
  typedef signed char Bsamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][COMPLEX][NR_TIMES_PER_BLOCK + 8][COMPLEX];
#elif NR_BITS == 16
  typedef __half      Asamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK][COMPLEX];
  typedef __half      Bsamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][COMPLEX][NR_TIMES_PER_BLOCK + 4][COMPLEX];
#endif
};


template <typename T> struct FetchData
{
  __device__ FetchData(unsigned loadRecv, unsigned loadPol, unsigned loadTime)
  :
    loadRecv(loadRecv), loadPol(loadPol), loadTime(loadTime), data({0})
  {
  }

  __device__ void load(const Samples samples, unsigned channel, unsigned time, unsigned firstReceiver, bool skipLoadCheck = NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0)
  {
    if (skipLoadCheck || firstReceiver + loadRecv < NR_RECEIVERS)
    {
#pragma unroll
      for (unsigned i = 0; i < sizeof(T) / sizeof(Sample); i++)
        memcpy((char *) &data + i * sizeof(Sample), &samples[firstReceiver + loadRecv][channel][time][loadTime + i][loadPol], sizeof(Sample));
    }
  }

  template <typename SharedData> __device__ void storeA(SharedData samples) const
  {
    //* ((T *) &samples[loadRecv][loadPol][loadTime][0]) = data;
    memcpy(&samples[loadRecv][loadPol][loadTime][0], &data, sizeof(T));
  }

  template <typename SharedData> __device__ void storeB(SharedData samples) const
  {
    //* ((T *) &samples[loadRecv][loadPol][0][loadTime][0]) = data;
    //* ((T *) &samples[loadRecv][loadPol][1][loadTime][0]) = conj_perm(data);
    T tmp = conj_perm(data);
    memcpy(&samples[loadRecv][loadPol][0][loadTime][0], &data, sizeof(T));
    memcpy(&samples[loadRecv][loadPol][1][loadTime][0], &tmp, sizeof(T));
  }

#if defined ASYNC_COPIES
  template <typename Asamples> __device__ void copyAsyncA(nvcuda::experimental::pipeline &pipe, Asamples dest, const Samples samples, unsigned channel, unsigned time, unsigned firstReceiver, bool skipLoadCheck = NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0)
  {
    if (skipLoadCheck || firstReceiver + loadRecv < NR_RECEIVERS)
    {
#pragma unroll
      for (unsigned i = 0; i < sizeof(T) / sizeof(Sample); i++)
        nvcuda::experimental::memcpy_async(* (Sample *) &dest[loadRecv][loadPol][loadTime + i][0], * (const Sample *) &samples[firstReceiver + loadRecv][channel][time][loadTime + i][loadPol], pipe);
    }
  }

  template<typename Bsamples> __device__ void copyAsyncB(nvcuda::experimental::pipeline &pipe, Bsamples dest, const Samples samples, unsigned channel, unsigned time, unsigned firstReceiver, bool skipLoadCheck = NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0)
  {
    if (skipLoadCheck || firstReceiver + loadRecv < NR_RECEIVERS)
    {
#pragma unroll
      for (unsigned i = 0; i < sizeof(T) / sizeof(Sample); i++)
        nvcuda::experimental::memcpy_async(* (Sample *) &dest[loadRecv][loadPol][0][loadTime + i][0], * (const Sample *) &samples[firstReceiver + loadRecv][channel][time][loadTime + i][loadPol], pipe);
    }
  }

  template<typename Bsamples> __device__ void fixB(Bsamples bSamples)
  {
    //* ((T *) &bSamples[loadRecv][loadPol][1][loadTime][0]) = conj_perm(* ((T *) &bSamples[loadRecv][loadPol][0][loadTime][0]));
    T tmp;
    memcpy(&tmp, &bSamples[loadRecv][loadPol][0][loadTime][0], sizeof(T));
    tmp = conj_perm(tmp);
    memcpy(&bSamples[loadRecv][loadPol][1][loadTime][0], &tmp, sizeof(T));
  }
#endif

  unsigned loadRecv, loadPol, loadTime;
  T        data;
};


__device__ inline int2 make_complex(int real, int imag)
{
  return make_int2(real, imag);
}


__device__ inline float2 make_complex(float real, float imag)
{
  return make_float2(real, imag);
}


// Compute x + y clamped to -INT_MAX .. INT_MAX
__device__ inline int add_sat(int x, int y)
{
  int out;
  asm("add.sat.s32 %0, %1, %2;" : "=r" (out) : "r" (x), "r" (y));
  // add.sat.s32 clamps to INT_MIN..INT_MAX, but we want -INT_MAX..INT_MAX
  // for symmetry.
  return max(out, INT_MIN + 1);
}


template <typename T> __device__ inline void accumVisibility(T &out, T value)
{
  /* Store an output value. Unlike the original ASTRON code, for xbgpu this
   * - conjugates the value because we want to store the other half
   *   (triangle) of the visibility matrix; and
   * - adds to the existing value (with saturation, if integer), to allow
   *   accumulation across multiple calls to the kernel.
   */
#if NR_BITS == 16
  out = make_complex(out.x + value.x, out.y - value.y);
#else
  out = make_complex(add_sat(out.x, value.x), add_sat(out.y, -value.y));
#endif
}


template <typename T> __device__ inline void storeVisibility(Visibilities visibilities, unsigned channel, unsigned baseline, unsigned recvY, unsigned recvX, unsigned tcY, unsigned tcX, unsigned polY, unsigned polX, bool skipCheckY, bool skipCheckX, T sumR, T sumI)
{
  if ((skipCheckX || recvX + tcX <= recvY + tcY) && (skipCheckY || recvY + tcY < NR_RECEIVERS))
  {
    accumVisibility(visibilities[channel][baseline + tcY * recvY + tcY * (tcY + 1) / 2 + tcX][polY][polX],
                    make_complex(sumR, sumI));
  }
}


__device__ inline void storeVisibilities(Visibilities visibilities, unsigned channel, unsigned firstReceiverY, unsigned firstReceiverX, unsigned recvYoffset, unsigned recvXoffset, unsigned y, unsigned x, bool skipCheckY, bool skipCheckX, const Sum &sum, ScratchSpace scratchSpace[], unsigned warp)
{
#if defined PORTABLE
 store_matrix_sync(&scratchSpace[warp][0][0][0][0].x, sum, NR_BITS == 4 ? 8 : 16, mem_row_major);
  __syncwarp();

#if 0
  if (threadIdx.x == 0)
    for (unsigned _y = 0; _y < 8; _y ++)
      for (unsigned pol_y = 0; pol_y < NR_POLARIZATIONS; pol_y ++)
        for (unsigned _x = 0; _x < 4; _x ++)
          for (unsigned pol_x = 0; pol_x < NR_POLARIZATIONS; pol_x ++)
            if (scratchSpace[warp][_y][pol_y][_x][pol_x],x != 0 || scratchSpace[warp][_y][pol_y][_x][pol_x].y != 0)
              printf("firstY=%u firstX=%u warp=%u y=%u x=%u _y=%u pol_y=%u _x=%u pol_x=%u val=(%f,%f)\n", firstReceiverY, firstReceiverX, warp, y, x, _y, pol_y, _x, pol_x, scratchSpace[warp][_y][pol_y][_x][pol_x].x, scratchSpace[warp][_y][pol_y][_x][pol_x].y);
#endif

#if NR_BITS == 4
  unsigned _y       = threadIdx.x >> 3;
  unsigned _x       = (threadIdx.x >> 2) & 1;
  unsigned polY     = (threadIdx.x >> 1) & 1;
  unsigned polX     = threadIdx.x & 1;
#elif NR_BITS == 8 || NR_BITS == 16
  unsigned _y       = threadIdx.x >> 2;
  unsigned _x       = threadIdx.x & 3;
#endif

  unsigned recvY    = firstReceiverY + recvYoffset + NR_RECEIVERS_PER_TCM_Y * y + _y;
  unsigned recvX    = firstReceiverX + recvXoffset + NR_RECEIVERS_PER_TCM_X * x + _x;
  unsigned baseline = (recvY * (recvY + 1) / 2) + recvX;

  if ((skipCheckX || recvX <= recvY) && (skipCheckY || recvY < NR_RECEIVERS))
#if NR_BITS == 4
    accumVisibility(visibilities[channel][baseline][polY][polX], scratchSpace[warp][_y][polY][_x][polX]);
#elif NR_BITS == 8 || NR_BITS == 16
    for (unsigned polY = 0; polY < NR_POLARIZATIONS; polY ++)
      for (unsigned polX = 0; polX < NR_POLARIZATIONS; polX ++)
        accumVisibility(visibilities[channel][baseline][polY][polX], scratchSpace[warp][_y][polY][_x][polX]);
#endif
#else
#if __CUDA_ARCH__ == 700 || (__CUDA_ARCH__ == 720 && NR_BITS == 16)
  unsigned recvY    = firstReceiverY + recvYoffset + NR_RECEIVERS_PER_TCM_Y * y + ((threadIdx.x >> 3) & 2) + (threadIdx.x & 4);
  unsigned recvX    = firstReceiverX + recvXoffset + NR_RECEIVERS_PER_TCM_X * x + ((threadIdx.x >> 2) & 2);
  unsigned polY     = threadIdx.x & 1;
  unsigned polX     = (threadIdx.x >> 1) & 1;
#elif (__CUDA_ARCH__ == 720 && NR_BITS == 8) || __CUDA_ARCH__ == 750 || __CUDA_ARCH__ == 800 || __CUDA_ARCH__ == 860
  unsigned recvY    = firstReceiverY + recvYoffset + NR_RECEIVERS_PER_TCM_Y * y + ((threadIdx.x >> 3) & 3);
  unsigned recvX    = firstReceiverX + recvXoffset + NR_RECEIVERS_PER_TCM_X * x + ((threadIdx.x >> 1) & 1);
  unsigned polY     = (threadIdx.x >> 2) & 1;
  unsigned polX     = threadIdx.x & 1;
#endif

  unsigned baseline = (recvY * (recvY + 1) / 2) + recvX;

#if __CUDA_ARCH__ == 700 || (__CUDA_ARCH__ == 720 && NR_BITS == 16)
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 0, polY, polX, skipCheckY, skipCheckX, sum.x[0], sum.x[1]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 1, polY, polX, skipCheckY, skipCheckX, sum.x[4], sum.x[5]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 1, 0, polY, polX, skipCheckY, skipCheckX, sum.x[2], sum.x[3]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 1, 1, polY, polX, skipCheckY, skipCheckX, sum.x[6], sum.x[7]);
#elif (__CUDA_ARCH__ == 720 && NR_BITS == 8) || __CUDA_ARCH__ == 750 || __CUDA_ARCH__ == 800 || __CUDA_ARCH__ == 860
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 0, polY, polX, skipCheckY, skipCheckX, sum.x[0], sum.x[1]);
#if NR_BITS == 8 || NR_BITS == 16
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 2, polY, polX, skipCheckY, skipCheckX, sum.x[4], sum.x[5]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 4, 0, polY, polX, skipCheckY, skipCheckX, sum.x[2], sum.x[3]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 4, 2, polY, polX, skipCheckY, skipCheckX, sum.x[6], sum.x[7]);
#endif
#endif
#endif
}


#define NR_WARPS 4

#if NR_RECEIVERS_PER_BLOCK == 64

template <bool fullTriangle> __device__ void doCorrelateTriangle(Visibilities visibilities, const Samples samples, unsigned firstReceiver, unsigned warp, unsigned tid, SharedData<>::Bsamples &bSamples, ScratchSpace scratchSpace[NR_WARPS])
{
  const unsigned nrFragmentsX = NR_BITS == 4 ? 12 : 6;
  const unsigned nrFragmentsY = nrFragmentsX / 2;
  Sum            sum[nrFragmentsX * nrFragmentsY];

  for (auto &s : sum)
    fill_fragment(s, 0);

  unsigned channel = blockIdx.y;

  const uchar2 offsets[] = {
    make_uchar2( 0,  0),
    make_uchar2( 0, 16),
    make_uchar2( 0, 40),
    make_uchar2(24, 40),
  };

  unsigned recvXoffset = offsets[warp].x;
  unsigned recvYoffset = offsets[warp].y;

  FetchData<int4> tmp0((tid >> 2)                             , (tid >> 1) & 1, 64 / NR_BITS * (tid & 1));
  FetchData<int4> tmp1((tid >> 2) + NR_RECEIVERS_PER_BLOCK / 2, (tid >> 1) & 1, 64 / NR_BITS * (tid & 1));

#if defined ASYNC_COPIES
  using namespace nvcuda::experimental;
  pipeline pipe;

  for (unsigned majorTime = 0; majorTime < READ_AHEAD; majorTime ++) {
    unsigned fetchBuffer = majorTime;

    tmp0.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorTime, firstReceiver, fullTriangle);
    tmp1.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorTime, firstReceiver, fullTriangle);

    pipe.commit();
  }
#else
  tmp0.load(samples, channel, 0, firstReceiver, fullTriangle);
  tmp1.load(samples, channel, 0, firstReceiver, fullTriangle);
#endif

  for (unsigned majorTime = 0; majorTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK; majorTime ++) {
    unsigned buffer = majorTime % NR_SHARED_BUFFERS;

#if !defined ASYNC_COPIES
    tmp0.storeB(bSamples[buffer]);
    tmp1.storeB(bSamples[buffer]);
#endif

    unsigned majorReadTime = majorTime + READ_AHEAD;

    if (majorReadTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK) {
#if defined ASYNC_COPIES
      unsigned fetchBuffer = (buffer + READ_AHEAD) % NR_SHARED_BUFFERS;

      tmp0.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorReadTime, firstReceiver, fullTriangle);
      tmp1.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorReadTime, firstReceiver, fullTriangle);
#else
      tmp0.load(samples, channel, majorReadTime, firstReceiver, fullTriangle);
      tmp1.load(samples, channel, majorReadTime, firstReceiver, fullTriangle);
#endif
    }

#if defined ASYNC_COPIES
    pipe.commit();
    pipe.wait_prior<READ_AHEAD>();

    tmp0.fixB(bSamples[buffer]);
    tmp1.fixB(bSamples[buffer]);
#endif

    __syncthreads();

#pragma unroll
    for (unsigned minorTime = 0; minorTime < NR_TIMES_PER_BLOCK; minorTime += ((NR_BITS) == 4 ? 16 : 8)) {
      Afrag aFrag[nrFragmentsY];
      Bfrag bFrag[nrFragmentsX];

      if (warp != 0) {
	for (unsigned y = 0; y < nrFragmentsY; y ++)
	  load_matrix_sync(aFrag[y], &bSamples[buffer][recvYoffset + NR_RECEIVERS_PER_TCM_Y * y][0][0][minorTime][0], sizeof(bSamples[0][0][0]) * 8 / NR_BITS);

	for (unsigned x = 0; x < nrFragmentsX; x ++)
	  load_matrix_sync(bFrag[x], &bSamples[buffer][recvXoffset + NR_RECEIVERS_PER_TCM_X * x][0][0][minorTime][0], sizeof(bSamples[0][0][0][0]) * 8 / NR_BITS);

	for (unsigned y = 0, i = 0; y < nrFragmentsY; y ++)
	  for (unsigned x = 0; x < nrFragmentsX; x ++, i ++)
	    mma_sync(sum[i], aFrag[y], bFrag[x], sum[i]);
      } else {
	for (unsigned z = 0, i = 0; z < 3; z ++) {
	  for (unsigned y = 0; y < (NR_BITS == 4 ? 4 : 2); y ++)
	    load_matrix_sync(aFrag[y], &bSamples[buffer][/*recvYoffset*/ 24 * z + NR_RECEIVERS_PER_TCM_Y * y][0][0][minorTime][0], sizeof(bSamples[0][0][0]) * 8 / NR_BITS);

	  for (unsigned x = 0; x < (NR_BITS == 4 ? 8 : 4); x ++)
	    load_matrix_sync(bFrag[x], &bSamples[buffer][/*recvXoffset*/ 24 * z + NR_RECEIVERS_PER_TCM_X * x][0][0][minorTime][0], sizeof(bSamples[0][0][0][0]) * 8 / NR_BITS);

	  for (unsigned y = 0; y < (NR_BITS == 4 ? 4 : 2); y ++)
	    for (unsigned x = 0; x < 2 + 2 * y; x ++, i ++)
	      mma_sync(sum[i], aFrag[y], bFrag[x], sum[i]);
	}
      }
    }
  }

#if defined PORTABLE
  __syncthreads();
#endif

  if (warp != 0)
    for (unsigned y = 0, i = 0; y < nrFragmentsY; y ++)
      for (unsigned x = 0; x < nrFragmentsX; x ++, i ++)
	storeVisibilities(visibilities, channel, firstReceiver, firstReceiver, recvYoffset, recvXoffset, y, x, fullTriangle, x < 2 * y + (NR_BITS == 4 ? 8 : 4), sum[i], scratchSpace, warp);
  else
    for (unsigned z = 0, i = 0; z < 3; z ++)
      for (unsigned y = 0; y < (NR_BITS == 4 ? 4 : 2); y ++)
	for (unsigned x = 0; x < 2 * y + 2; x ++, i ++)
	  storeVisibilities(visibilities, channel, firstReceiver, firstReceiver, 24 * z, 24 * z, y, x, fullTriangle, x < 2 * y, sum[i], scratchSpace, warp);
}

#endif


template <unsigned nrFragmentsY, bool skipLoadYcheck, bool skipLoadXcheck, bool skipStoreYcheck, bool skipStoreXcheck> __device__ void doCorrelateRectangle(Visibilities visibilities, const Samples samples, unsigned firstReceiverY, unsigned firstReceiverX, SharedData<>::Asamples &aSamples, SharedData<NR_RECEIVERS_PER_BLOCK == 64 ? 32 : NR_RECEIVERS_PER_BLOCK>::Bsamples &bSamples, ScratchSpace scratchSpace[NR_WARPS])
{
  const unsigned nrFragmentsX = NR_RECEIVERS_PER_BLOCK / NR_RECEIVERS_PER_TCM_X / 2 / (NR_RECEIVERS_PER_BLOCK == 64 ? 2 : 1);

  Sum sum[nrFragmentsY][nrFragmentsX];

  for (unsigned y = 0; y < nrFragmentsY; y ++)
    for (unsigned x = 0; x < nrFragmentsX; x ++)
      fill_fragment(sum[y][x], 0);

  unsigned tid     = warpSize * (blockDim.y * threadIdx.z + threadIdx.y) + threadIdx.x;
  unsigned channel = blockIdx.y;

  unsigned recvXoffset = nrFragmentsX * NR_RECEIVERS_PER_TCM_X * threadIdx.y;
  unsigned recvYoffset = nrFragmentsY * NR_RECEIVERS_PER_TCM_Y * threadIdx.z;

  FetchData<int4> tmpY0((tid >> 2)     , (tid >> 1) & 1, 64 / NR_BITS * (tid & 1));
  FetchData<int4> tmpX0((tid >> 2)     , (tid >> 1) & 1, 64 / NR_BITS * (tid & 1));
#if NR_RECEIVERS_PER_BLOCK == 48
  FetchData<int2> tmpY1((tid >> 3) + 32, (tid >> 2) & 1, 32 / NR_BITS * (tid & 3));
  FetchData<int2> tmpX1((tid >> 3) + 32, (tid >> 2) & 1, 32 / NR_BITS * (tid & 3));
#elif NR_RECEIVERS_PER_BLOCK == 64
  FetchData<int4> tmpY1((tid >> 2) + 32, (tid >> 1) & 1, 64 / NR_BITS * (tid & 1));
#endif

#if defined ASYNC_COPIES
  using namespace nvcuda::experimental;
  pipeline pipe;

  for (unsigned majorTime = 0; majorTime < READ_AHEAD; majorTime ++) {
    unsigned fetchBuffer = majorTime;

    tmpY0.copyAsyncA(pipe, aSamples[fetchBuffer], samples, channel, majorTime, firstReceiverY, skipLoadYcheck);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
    tmpY1.copyAsyncA(pipe, aSamples[fetchBuffer], samples, channel, majorTime, firstReceiverY, skipLoadYcheck);
#endif
    tmpX0.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorTime, firstReceiverX, skipLoadXcheck);
#if NR_RECEIVERS_PER_BLOCK == 48
    tmpX1.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorTime, firstReceiverX, skipLoadXcheck);
#endif

    pipe.commit();
  }
#else
  tmpY0.load(samples, channel, 0, firstReceiverY, skipLoadYcheck);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
  tmpY1.load(samples, channel, 0, firstReceiverY, skipLoadYcheck);
#endif
  tmpX0.load(samples, channel, 0, firstReceiverX, skipLoadXcheck);
#if NR_RECEIVERS_PER_BLOCK == 48
  tmpX1.load(samples, channel, 0, firstReceiverX, skipLoadXcheck);
#endif
#endif

  for (unsigned majorTime = 0; majorTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK; majorTime ++) {
    unsigned buffer = majorTime % NR_SHARED_BUFFERS;

#if !defined ASYNC_COPIES
    tmpY0.storeA(aSamples[buffer]);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
    tmpY1.storeA(aSamples[buffer]);
#endif
    tmpX0.storeB(bSamples[buffer]);
#if NR_RECEIVERS_PER_BLOCK == 48
    tmpX1.storeB(bSamples[buffer]);
#endif
#endif

    unsigned majorReadTime = majorTime + READ_AHEAD;

    if (majorReadTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK) {
#if defined ASYNC_COPIES
      unsigned fetchBuffer = (buffer + READ_AHEAD) % NR_SHARED_BUFFERS;

      tmpY0.copyAsyncA(pipe, aSamples[fetchBuffer], samples, channel, majorReadTime, firstReceiverY, skipLoadYcheck);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
      tmpY1.copyAsyncA(pipe, aSamples[fetchBuffer], samples, channel, majorReadTime, firstReceiverY, skipLoadYcheck);
#endif
      tmpX0.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorReadTime, firstReceiverX, skipLoadXcheck);
#if NR_RECEIVERS_PER_BLOCK == 48
      tmpX1.copyAsyncB(pipe, bSamples[fetchBuffer], samples, channel, majorReadTime, firstReceiverX, skipLoadXcheck);
#endif
#else
      tmpY0.load(samples, channel, majorReadTime, firstReceiverY, skipLoadYcheck);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
      tmpY1.load(samples, channel, majorReadTime, firstReceiverY, skipLoadYcheck);
#endif
      tmpX0.load(samples, channel, majorReadTime, firstReceiverX, skipLoadXcheck);
#if NR_RECEIVERS_PER_BLOCK == 48
      tmpX1.load(samples, channel, majorReadTime, firstReceiverX, skipLoadXcheck);
#endif
#endif
    }

#if defined ASYNC_COPIES
    pipe.commit();
    pipe.wait_prior<READ_AHEAD>();

    tmpX0.fixB(bSamples[buffer]);
#if NR_RECEIVERS_PER_BLOCK == 48
    tmpX1.fixB(bSamples[buffer]);
#endif
#endif

    __syncthreads();

#pragma unroll
    for (unsigned minorTime = 0; minorTime < NR_TIMES_PER_BLOCK; minorTime += ((NR_BITS) == 4 ? 16 : 8)) {
      Afrag aFrag[nrFragmentsY];
      Bfrag bFrag[nrFragmentsX];

      for (unsigned y = 0; y < nrFragmentsY; y ++)
	load_matrix_sync(aFrag[y], &aSamples[buffer][recvYoffset + NR_RECEIVERS_PER_TCM_Y * y][0][minorTime][0], sizeof(aSamples[0][0][0]) * 8 / NR_BITS);

      for (unsigned x = 0; x < nrFragmentsX; x ++)
	load_matrix_sync(bFrag[x], &bSamples[buffer][recvXoffset + NR_RECEIVERS_PER_TCM_X * x][0][0][minorTime][0], sizeof(bSamples[0][0][0][0]) * 8 / NR_BITS);

      for (unsigned y = 0; y < nrFragmentsY; y ++)
	for (unsigned x = 0; x < nrFragmentsX; x ++)
	  mma_sync(sum[y][x], aFrag[y], bFrag[x], sum[y][x]);
    }
  }

#if 0
  for (unsigned y = 0; y < nrFragmentsY; y ++)
    for (unsigned x = 0; x < nrFragmentsX; x ++)
      for (unsigned i = 0; i < sum[0][0].num_storage_elements; i ++)
	if (sum[y][x].x[i] != 0)
#if NR_BITS == 4 || NR_BITS == 8
	  printf("blockIdx=(%d,%d,%d) tid=%u y=%u x=%u i=%u v=%d\n", blockIdx.x, blockIdx.y, blockIdx.z, tid, y, x, i, sum[y][x].x[i]);
#else
	  printf("blockIdx=(%d,%d,%d) tid=%u y=%u x=%u i=%u v=%f\n", blockIdx.x, blockIdx.y, blockIdx.z, tid, y, x, i, sum[y][x].x[i]);
#endif
#endif

#if defined PORTABLE
  __syncthreads();
#endif

  for (unsigned y = 0; y < nrFragmentsY; y ++)
    for (unsigned x = 0; x < nrFragmentsX; x ++)
      storeVisibilities(visibilities, channel, firstReceiverY, firstReceiverX, recvYoffset, recvXoffset, y, x, skipStoreYcheck, skipStoreXcheck, sum[y][x], scratchSpace, tid / warpSize);
}


extern "C" __global__
__launch_bounds__(NR_WARPS * 32, NR_RECEIVERS_PER_BLOCK == 32 ? 4 : 2)
void correlate(Visibilities visibilities, const Samples samples)
{
  const unsigned nrFragmentsY = NR_RECEIVERS_PER_BLOCK / NR_RECEIVERS_PER_TCM_Y / 2;

  unsigned block = blockIdx.x;

#if NR_RECEIVERS_PER_BLOCK == 32 || NR_RECEIVERS_PER_BLOCK == 48
  unsigned blockY = (unsigned) (sqrtf(8 * block + 1) - .99999f) / 2;
  unsigned blockX = block - blockY * (blockY + 1) / 2;
  unsigned firstReceiverX = blockX * NR_RECEIVERS_PER_BLOCK;
#elif NR_RECEIVERS_PER_BLOCK == 64
  unsigned blockY = (unsigned) sqrtf(block);
  unsigned blockX = block - blockY * blockY;
  unsigned firstReceiverX = blockX * (NR_RECEIVERS_PER_BLOCK / 2);
#endif
  unsigned firstReceiverY = blockY * NR_RECEIVERS_PER_BLOCK;

  union shared {
    struct {
      SharedData<>::Asamples aSamples;
      SharedData<NR_RECEIVERS_PER_BLOCK == 64 ? 32 : NR_RECEIVERS_PER_BLOCK>::Bsamples bSamples;
    } rectangle;
    struct {
      SharedData<>::Bsamples samples;
    } triangle;
    ScratchSpace scratchSpace[NR_WARPS];
  };

  // the following hack is necessary to run the correlator in the OpenCL environment,
  // as the maximum local memory size is 48K - 16 bytes.  Due to padding in bSamples,
  // the last 16 bytes are not used, so allocate 16 fewer bytes.
  __shared__ char rawbuffer[sizeof(union shared) - 16] __attribute__((aligned(16)));
  union shared &u = (union shared &) rawbuffer;

  if (firstReceiverX == firstReceiverY)
#if NR_RECEIVERS_PER_BLOCK == 32 || NR_RECEIVERS_PER_BLOCK == 48
    doCorrelateRectangle<nrFragmentsY, NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0, NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0, NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0, false>(visibilities, samples, firstReceiverY, firstReceiverX, u.rectangle.aSamples, u.rectangle.bSamples, u.scratchSpace);
#elif NR_RECEIVERS_PER_BLOCK == 64
    if (NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK != 0 && (NR_RECEIVERS < NR_RECEIVERS_PER_BLOCK || firstReceiverX >= NR_RECEIVERS / NR_RECEIVERS_PER_BLOCK * NR_RECEIVERS_PER_BLOCK))
      doCorrelateTriangle<false>(visibilities, samples, firstReceiverX, 2 * threadIdx.z + threadIdx.y, 64 * threadIdx.z + 32 * threadIdx.y + threadIdx.x, u.triangle.samples, u.scratchSpace);
    else
      doCorrelateTriangle<true>(visibilities, samples, firstReceiverX, 2 * threadIdx.z + threadIdx.y, 64 * threadIdx.z + 32 * threadIdx.y + threadIdx.x, u.triangle.samples, u.scratchSpace);
#endif
#if NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK != 0
  else if (NR_RECEIVERS < NR_RECEIVERS_PER_BLOCK || firstReceiverY >= NR_RECEIVERS / NR_RECEIVERS_PER_BLOCK * NR_RECEIVERS_PER_BLOCK)
    doCorrelateRectangle<(NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK + 2 * NR_RECEIVERS_PER_TCM_Y - 1) / NR_RECEIVERS_PER_TCM_Y / 2, false, true, NR_RECEIVERS % (2 * NR_RECEIVERS_PER_TCM_Y) == 0, true>(visibilities, samples, firstReceiverY, firstReceiverX, u.rectangle.aSamples, u.rectangle.bSamples, u.scratchSpace);
#endif
  else
    doCorrelateRectangle<nrFragmentsY, true, true, true, true>(visibilities, samples, firstReceiverY, firstReceiverX, u.rectangle.aSamples, u.rectangle.bSamples, u.scratchSpace);
}

} // extern "C++"
