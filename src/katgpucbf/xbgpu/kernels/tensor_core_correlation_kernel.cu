#include "hip/hip_runtime.h"
/*    Original license:
 *
 *    Copyright 2021 ASTRON
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 */

/* This code is based on
 * https://git.astron.nl/RD/tensor-core-correlator/-/blob/83abdcc/libtcc/TCCorrelator.cu
 *
 * See https://developer.nvidia.com/gtc/2019/video/s9306 for a high-level overview.
 * Lower-level details are in the doc/xbgpu.tcc.rst (and built by Sphinx with
 * the rest of the documentation).
 *
 * It has been modified by SARAO:
 * - Wrap the file in extern "C++" to make it work with PyCUDA (see below)
 * - Add results to the output instead of overwriting, to allow accumulation
 *   across multiple calls; results use 64-bit integers to avoid overflow.
 * - Conjugate the output, to provide the other triangle of the visibility
 *   matrix.
 * - Take the input axes in a different order.
 * - Remove the asynchronous copy code (it would not have worked well with
 *   the previous point).
 * - Restore the type-punning that had been replaced by memcpy. It turns out
 *   nvcc implements memcpy a byte at a time.
 * - Guarantee 32-byte alignment of the shared data (required by
 *   load_matrix_sync / store_matrix_sync).
 * - Parallelise over multiple problem instances.
 * - Write the output
 * - Remove trailing whitespace.
 *
 * SARAO's modification is licenced as follows:
 *******************************************************************************
 * Copyright (c) 2020-2021, National Research Foundation (SARAO)
 *
 * Licensed under the BSD 3-Clause License (the "License"); you may not use
 * this file except in compliance with the License. You may obtain a copy
 * of the License at
 *
 *   https://opensource.org/licenses/BSD-3-Clause
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ******************************************************************************/

/* PyCUDA wraps the whole file in 'extern "C"', but most of the code expects
 * C++ linkage. So we wrap the whole original file in 'extern "C++"' to cancel
 * that out.
 *
 * When this code gets closer to production, the suggested fix is to modify the
 * accel.build() and context.compile() functions in katsdpsigproc to take a
 * no_extern_c flag as these ones are the methods that will call the
 * pycuda.compiler.SourceModule(...) constructor.
 */
extern "C++" {

#include <mma.h>

#define NR_BASELINES		(NR_RECEIVERS * (NR_RECEIVERS + 1) / 2)
#define ALIGN(A,N)		(((A)+(N)-1)/(N)*(N))

#define NR_TIMES_PER_BLOCK	(128 / (NR_BITS))
#define NR_RECEIVERS_PER_TCM_X	((NR_BITS) == 4 ? 2 : 4)
#define NR_RECEIVERS_PER_TCM_Y	((NR_BITS) == 4 ? 4 : 8)

#define COMPLEX			2

#if __CUDA_ARCH__ < (NR_BITS == 4 ? 730 : NR_BITS == 8 ? 720 : NR_BITS == 16 ? 700 : 0)
#error this architecture has no suitable tensor cores
#endif

#if __CUDA_ARCH__ != 700 && __CUDA_ARCH__ != 720 && __CUDA_ARCH__ != 750 && __CUDA_ARCH__ != 800 && __CUDA_ARCH__ != 860
#define PORTABLE // unknown architecture -> write visibilities in portable way (via shared memory)
#endif

#if NR_RECEIVERS_PER_BLOCK != 32 && NR_RECEIVERS_PER_BLOCK != 48 && NR_RECEIVERS_PER_BLOCK != 64
#error unsupported NR_RECEIVERS_PER_BLOCK
#endif

#if NR_SAMPLES_PER_CHANNEL % NR_TIMES_PER_BLOCK != 0
#error NR_SAMPLES_PER_CHANNEL should be a multiple of NR_TIMES_PER_BLOCK
#endif

#define MIN(A,B) ((A)<(B)?(A):(B))


using namespace nvcuda::wmma;

#if NR_BITS == 4
typedef char    Sample;
typedef long2   Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];
#elif NR_BITS == 8
typedef char2   Sample;
typedef long2   Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];
#elif NR_BITS == 16
typedef __half2 Sample;
typedef float2  Visibilities[NR_CHANNELS][NR_BASELINES][NR_POLARIZATIONS][NR_POLARIZATIONS];
#endif
typedef Sample Samples[NR_RECEIVERS][NR_CHANNELS][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_TIMES_PER_BLOCK][NR_POLARIZATIONS];

#if NR_BITS == 4
typedef fragment<matrix_a, 8, 8, 32, experimental::precision::s4, row_major> Afrag;
typedef fragment<matrix_b, 8, 8, 32, experimental::precision::s4, col_major> Bfrag;
typedef fragment<accumulator, 8, 8, 32, int>                                 Sum;
#elif NR_BITS == 8
typedef fragment<matrix_a, 16, 16, 16, signed char, row_major>               Afrag;
typedef fragment<matrix_b, 16, 16, 16, signed char, col_major>               Bfrag;
typedef fragment<accumulator, 16, 16, 16, int>                               Sum;
#elif NR_BITS == 16
typedef fragment<matrix_a, 16, 16, 16, __half, row_major>                    Afrag;
typedef fragment<matrix_b, 16, 16, 16, __half, col_major>                    Bfrag;
typedef fragment<accumulator, 16, 16, 16, float>                             Sum;
#endif


#if NR_BITS == 4
typedef int2   ScratchSpace[4][NR_POLARIZATIONS][2][NR_POLARIZATIONS];
#elif NR_BITS == 8
typedef int2   ScratchSpace[8][NR_POLARIZATIONS][4][NR_POLARIZATIONS];
#elif NR_BITS == 16
typedef float2 ScratchSpace[8][NR_POLARIZATIONS][4][NR_POLARIZATIONS];
#endif


__device__ inline int conj_perm(int v)
{
#if NR_BITS == 4
  //return ((v & 0x0F0F0F0F) << 4) | (__vnegss4(v >> 4) & 0x0F0F0F0F);
  return ((v & 0x0F0F0F0F) << 4) | ((0xF0F0F0F0 - ((v >> 4) & 0x0F0F0F0F)) & 0x0F0F0F0F);
#elif NR_BITS == 8
  //return __byte_perm(v, __vnegss4(v), 0x2705);
  return __byte_perm(v, 0x00FF00FF - (v & 0xFF00FF00), 0x2705);
#elif NR_BITS == 16
  return __byte_perm(v ^ 0x80000000, v, 0x1032);
#endif
}


__device__ inline int2 conj_perm(int2 v)
{
  return make_int2(conj_perm(v.x), conj_perm(v.y));
}


__device__ inline int4 conj_perm(int4 v)
{
  return make_int4(conj_perm(v.x), conj_perm(v.y), conj_perm(v.z), conj_perm(v.w));
}


#define READ_AHEAD        1
#define NR_SHARED_BUFFERS 2


template <unsigned nrReceiversPerBlock = NR_RECEIVERS_PER_BLOCK> struct SharedData
{
#if NR_BITS == 4
  typedef char        Asamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK][1];
  typedef char        Bsamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][COMPLEX][NR_TIMES_PER_BLOCK + 16][1];
#elif NR_BITS == 8
  typedef signed char Asamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK][COMPLEX];
  typedef signed char Bsamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][COMPLEX][NR_TIMES_PER_BLOCK + 8][COMPLEX];
#elif NR_BITS == 16
  typedef __half      Asamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK][COMPLEX];
  typedef __half      Bsamples[NR_SHARED_BUFFERS][nrReceiversPerBlock][NR_POLARIZATIONS][COMPLEX][NR_TIMES_PER_BLOCK + 4][COMPLEX];
#endif
};


template <typename T> struct FetchData
{
  __device__ FetchData(unsigned loadRecv, unsigned loadTime)
  :
    loadRecv(loadRecv), loadTime(loadTime), data({0})
  {
  }

  __device__ void load(const Samples samples, unsigned channel, unsigned time, unsigned firstReceiver, bool skipLoadCheck = NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0)
  {
    if (skipLoadCheck || firstReceiver + loadRecv < NR_RECEIVERS)
    {
      data = * (T *) &samples[firstReceiver + loadRecv][channel][time][loadTime][0];
      // The above is undefined behaviour in C++ (type punning), but the
      // well-defined memcpy below has poor performance (copies one byte at a time).
      // memcpy(&data, &samples[firstReceiver + loadRecv][channel][time][loadTime][0], sizeof(T));
    }
  }

  template <typename SharedData> __device__ void storeA(SharedData samples) const
  {
#pragma unroll
    for (unsigned i = 0; i < sizeof(T) / sizeof(Sample); i++)
      *(Sample *) &samples[loadRecv][i & 1][loadTime + (i >> 1)][0] = ((const Sample *) &data)[i];
  }

  template <typename SharedData> __device__ void storeB(SharedData samples) const
  {
    //* ((T *) &samples[loadRecv][loadPol][0][loadTime][0]) = data;
    //* ((T *) &samples[loadRecv][loadPol][1][loadTime][0]) = conj_perm(data);
    T tmp = conj_perm(data);
#pragma unroll
    for (unsigned i = 0; i < sizeof(T) / sizeof(Sample); i++)
    {
      unsigned time = loadTime + (i >> 1);
      *(Sample *) &samples[loadRecv][i & 1][0][time][0] = ((const Sample *) &data)[i];
      *(Sample *) &samples[loadRecv][i & 1][1][time][0] = ((const Sample *) &tmp)[i];
    }
  }

  unsigned loadRecv, loadPol, loadTime;
  T        data;
};


__device__ inline int2 make_complex(int real, int imag)
{
  return make_int2(real, imag);
}


__device__ inline float2 make_complex(float real, float imag)
{
  return make_float2(real, imag);
}


__device__ inline long2 make_complex(long real, long imag)
{
  return make_long2(real, imag);
}


template <typename T, typename V> __device__ inline void accumVisibility(T &out, V value)
{
  /* Store an output value. Unlike the original ASTRON code, for xbgpu this
   * - conjugates the value because we want to store the other half
   *   (triangle) of the visibility matrix; and
   * - adds to the existing value (with saturation, if integer), to allow
   *   accumulation across multiple calls to the kernel.
   */
  out = make_complex(out.x + value.x, out.y - value.y);
}


template <typename T> __device__ inline void storeVisibility(Visibilities visibilities, unsigned channel, unsigned baseline, unsigned recvY, unsigned recvX, unsigned tcY, unsigned tcX, unsigned polY, unsigned polX, bool skipCheckY, bool skipCheckX, T sumR, T sumI)
{
  if ((skipCheckX || recvX + tcX <= recvY + tcY) && (skipCheckY || recvY + tcY < NR_RECEIVERS))
  {
    accumVisibility(visibilities[channel][baseline + tcY * recvY + tcY * (tcY + 1) / 2 + tcX][polY][polX],
                    make_complex(sumR, sumI));
  }
}


__device__ inline void storeVisibilities(Visibilities visibilities, unsigned channel, unsigned firstReceiverY, unsigned firstReceiverX, unsigned recvYoffset, unsigned recvXoffset, unsigned y, unsigned x, bool skipCheckY, bool skipCheckX, const Sum &sum, ScratchSpace scratchSpace[], unsigned warp)
{
#if defined PORTABLE
 store_matrix_sync(&scratchSpace[warp][0][0][0][0].x, sum, NR_BITS == 4 ? 8 : 16, mem_row_major);
  __syncwarp();

#if 0
  if (threadIdx.x == 0)
    for (unsigned _y = 0; _y < 8; _y ++)
      for (unsigned pol_y = 0; pol_y < NR_POLARIZATIONS; pol_y ++)
        for (unsigned _x = 0; _x < 4; _x ++)
          for (unsigned pol_x = 0; pol_x < NR_POLARIZATIONS; pol_x ++)
            if (scratchSpace[warp][_y][pol_y][_x][pol_x],x != 0 || scratchSpace[warp][_y][pol_y][_x][pol_x].y != 0)
              printf("firstY=%u firstX=%u warp=%u y=%u x=%u _y=%u pol_y=%u _x=%u pol_x=%u val=(%f,%f)\n", firstReceiverY, firstReceiverX, warp, y, x, _y, pol_y, _x, pol_x, scratchSpace[warp][_y][pol_y][_x][pol_x].x, scratchSpace[warp][_y][pol_y][_x][pol_x].y);
#endif

#if NR_BITS == 4
  unsigned _y       = threadIdx.x >> 3;
  unsigned _x       = (threadIdx.x >> 2) & 1;
  unsigned polY     = (threadIdx.x >> 1) & 1;
  unsigned polX     = threadIdx.x & 1;
#elif NR_BITS == 8 || NR_BITS == 16
  unsigned _y       = threadIdx.x >> 2;
  unsigned _x       = threadIdx.x & 3;
#endif

  unsigned recvY    = firstReceiverY + recvYoffset + NR_RECEIVERS_PER_TCM_Y * y + _y;
  unsigned recvX    = firstReceiverX + recvXoffset + NR_RECEIVERS_PER_TCM_X * x + _x;
  unsigned baseline = (recvY * (recvY + 1) / 2) + recvX;

  if ((skipCheckX || recvX <= recvY) && (skipCheckY || recvY < NR_RECEIVERS))
#if NR_BITS == 4
    accumVisibility(visibilities[channel][baseline][polY][polX], scratchSpace[warp][_y][polY][_x][polX]);
#elif NR_BITS == 8 || NR_BITS == 16
    for (unsigned polY = 0; polY < NR_POLARIZATIONS; polY ++)
      for (unsigned polX = 0; polX < NR_POLARIZATIONS; polX ++)
        accumVisibility(visibilities[channel][baseline][polY][polX], scratchSpace[warp][_y][polY][_x][polX]);
#endif
#else
#if __CUDA_ARCH__ == 700 || (__CUDA_ARCH__ == 720 && NR_BITS == 16)
  unsigned recvY    = firstReceiverY + recvYoffset + NR_RECEIVERS_PER_TCM_Y * y + ((threadIdx.x >> 3) & 2) + (threadIdx.x & 4);
  unsigned recvX    = firstReceiverX + recvXoffset + NR_RECEIVERS_PER_TCM_X * x + ((threadIdx.x >> 2) & 2);
  unsigned polY     = threadIdx.x & 1;
  unsigned polX     = (threadIdx.x >> 1) & 1;
#elif (__CUDA_ARCH__ == 720 && NR_BITS == 8) || __CUDA_ARCH__ == 750 || __CUDA_ARCH__ == 800 || __CUDA_ARCH__ == 860
  unsigned recvY    = firstReceiverY + recvYoffset + NR_RECEIVERS_PER_TCM_Y * y + ((threadIdx.x >> 3) & 3);
  unsigned recvX    = firstReceiverX + recvXoffset + NR_RECEIVERS_PER_TCM_X * x + ((threadIdx.x >> 1) & 1);
  unsigned polY     = (threadIdx.x >> 2) & 1;
  unsigned polX     = threadIdx.x & 1;
#endif

  unsigned baseline = (recvY * (recvY + 1) / 2) + recvX;

#if __CUDA_ARCH__ == 700 || (__CUDA_ARCH__ == 720 && NR_BITS == 16)
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 0, polY, polX, skipCheckY, skipCheckX, sum.x[0], sum.x[1]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 1, polY, polX, skipCheckY, skipCheckX, sum.x[4], sum.x[5]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 1, 0, polY, polX, skipCheckY, skipCheckX, sum.x[2], sum.x[3]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 1, 1, polY, polX, skipCheckY, skipCheckX, sum.x[6], sum.x[7]);
#elif (__CUDA_ARCH__ == 720 && NR_BITS == 8) || __CUDA_ARCH__ == 750 || __CUDA_ARCH__ == 800 || __CUDA_ARCH__ == 860
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 0, polY, polX, skipCheckY, skipCheckX, sum.x[0], sum.x[1]);
#if NR_BITS == 8 || NR_BITS == 16
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 0, 2, polY, polX, skipCheckY, skipCheckX, sum.x[4], sum.x[5]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 4, 0, polY, polX, skipCheckY, skipCheckX, sum.x[2], sum.x[3]);
  storeVisibility(visibilities, channel, baseline, recvY, recvX, 4, 2, polY, polX, skipCheckY, skipCheckX, sum.x[6], sum.x[7]);
#endif
#endif
#endif
}


#define NR_WARPS 4

#if NR_RECEIVERS_PER_BLOCK == 64

template <bool fullTriangle> __device__ void doCorrelateTriangle(Visibilities visibilities, const Samples samples, unsigned firstReceiver, unsigned warp, unsigned tid, SharedData<>::Bsamples &bSamples, ScratchSpace scratchSpace[NR_WARPS])
{
  const unsigned nrFragmentsX = NR_BITS == 4 ? 12 : 6;
  const unsigned nrFragmentsY = nrFragmentsX / 2;
  Sum            sum[nrFragmentsX * nrFragmentsY];

  for (auto &s : sum)
    fill_fragment(s, 0);

  unsigned channel = blockIdx.y;

  const uchar2 offsets[] = {
    make_uchar2( 0,  0),
    make_uchar2( 0, 16),
    make_uchar2( 0, 40),
    make_uchar2(24, 40),
  };

  unsigned recvXoffset = offsets[warp].x;
  unsigned recvYoffset = offsets[warp].y;

  FetchData<int4> tmp0((tid >> 2)                             , 32 / NR_BITS * (tid & 3));
  FetchData<int4> tmp1((tid >> 2) + NR_RECEIVERS_PER_BLOCK / 2, 32 / NR_BITS * (tid & 3));

  tmp0.load(samples, channel, 0, firstReceiver, fullTriangle);
  tmp1.load(samples, channel, 0, firstReceiver, fullTriangle);

  for (unsigned majorTime = 0; majorTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK; majorTime ++) {
    unsigned buffer = majorTime % NR_SHARED_BUFFERS;

    tmp0.storeB(bSamples[buffer]);
    tmp1.storeB(bSamples[buffer]);

    unsigned majorReadTime = majorTime + READ_AHEAD;

    if (majorReadTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK) {
      tmp0.load(samples, channel, majorReadTime, firstReceiver, fullTriangle);
      tmp1.load(samples, channel, majorReadTime, firstReceiver, fullTriangle);
    }

    __syncthreads();

#pragma unroll
    for (unsigned minorTime = 0; minorTime < NR_TIMES_PER_BLOCK; minorTime += ((NR_BITS) == 4 ? 16 : 8)) {
      Afrag aFrag[nrFragmentsY];
      Bfrag bFrag[nrFragmentsX];

      if (warp != 0) {
	for (unsigned y = 0; y < nrFragmentsY; y ++)
	  load_matrix_sync(aFrag[y], &bSamples[buffer][recvYoffset + NR_RECEIVERS_PER_TCM_Y * y][0][0][minorTime][0], sizeof(bSamples[0][0][0]) * 8 / NR_BITS);

	for (unsigned x = 0; x < nrFragmentsX; x ++)
	  load_matrix_sync(bFrag[x], &bSamples[buffer][recvXoffset + NR_RECEIVERS_PER_TCM_X * x][0][0][minorTime][0], sizeof(bSamples[0][0][0][0]) * 8 / NR_BITS);

	for (unsigned y = 0, i = 0; y < nrFragmentsY; y ++)
	  for (unsigned x = 0; x < nrFragmentsX; x ++, i ++)
	    mma_sync(sum[i], aFrag[y], bFrag[x], sum[i]);
      } else {
	for (unsigned z = 0, i = 0; z < 3; z ++) {
	  for (unsigned y = 0; y < (NR_BITS == 4 ? 4 : 2); y ++)
	    load_matrix_sync(aFrag[y], &bSamples[buffer][/*recvYoffset*/ 24 * z + NR_RECEIVERS_PER_TCM_Y * y][0][0][minorTime][0], sizeof(bSamples[0][0][0]) * 8 / NR_BITS);

	  for (unsigned x = 0; x < (NR_BITS == 4 ? 8 : 4); x ++)
	    load_matrix_sync(bFrag[x], &bSamples[buffer][/*recvXoffset*/ 24 * z + NR_RECEIVERS_PER_TCM_X * x][0][0][minorTime][0], sizeof(bSamples[0][0][0][0]) * 8 / NR_BITS);

	  for (unsigned y = 0; y < (NR_BITS == 4 ? 4 : 2); y ++)
	    for (unsigned x = 0; x < 2 + 2 * y; x ++, i ++)
	      mma_sync(sum[i], aFrag[y], bFrag[x], sum[i]);
	}
      }
    }
  }

#if defined PORTABLE
  __syncthreads();
#endif

  if (warp != 0)
    for (unsigned y = 0, i = 0; y < nrFragmentsY; y ++)
      for (unsigned x = 0; x < nrFragmentsX; x ++, i ++)
	storeVisibilities(visibilities, channel, firstReceiver, firstReceiver, recvYoffset, recvXoffset, y, x, fullTriangle, x < 2 * y + (NR_BITS == 4 ? 8 : 4), sum[i], scratchSpace, warp);
  else
    for (unsigned z = 0, i = 0; z < 3; z ++)
      for (unsigned y = 0; y < (NR_BITS == 4 ? 4 : 2); y ++)
	for (unsigned x = 0; x < 2 * y + 2; x ++, i ++)
	  storeVisibilities(visibilities, channel, firstReceiver, firstReceiver, 24 * z, 24 * z, y, x, fullTriangle, x < 2 * y, sum[i], scratchSpace, warp);
}

#endif


template <unsigned nrFragmentsY, bool skipLoadYcheck, bool skipLoadXcheck, bool skipStoreYcheck, bool skipStoreXcheck> __device__ void doCorrelateRectangle(Visibilities visibilities, const Samples samples, unsigned firstReceiverY, unsigned firstReceiverX, SharedData<>::Asamples &aSamples, SharedData<NR_RECEIVERS_PER_BLOCK == 64 ? 32 : NR_RECEIVERS_PER_BLOCK>::Bsamples &bSamples, ScratchSpace scratchSpace[NR_WARPS])
{
  const unsigned nrFragmentsX = NR_RECEIVERS_PER_BLOCK / NR_RECEIVERS_PER_TCM_X / 2 / (NR_RECEIVERS_PER_BLOCK == 64 ? 2 : 1);

  Sum sum[nrFragmentsY][nrFragmentsX];

  for (unsigned y = 0; y < nrFragmentsY; y ++)
    for (unsigned x = 0; x < nrFragmentsX; x ++)
      fill_fragment(sum[y][x], 0);

  unsigned tid     = warpSize * (blockDim.y * threadIdx.z + threadIdx.y) + threadIdx.x;
  unsigned channel = blockIdx.y;

  unsigned recvXoffset = nrFragmentsX * NR_RECEIVERS_PER_TCM_X * threadIdx.y;
  unsigned recvYoffset = nrFragmentsY * NR_RECEIVERS_PER_TCM_Y * threadIdx.z;

  FetchData<int4> tmpY0((tid >> 2)     , 32 / NR_BITS * (tid & 3));
  FetchData<int4> tmpX0((tid >> 2)     , 32 / NR_BITS * (tid & 3));
#if NR_RECEIVERS_PER_BLOCK == 48
  FetchData<int2> tmpY1((tid >> 3) + 32, 16 / NR_BITS * (tid & 7));
  FetchData<int2> tmpX1((tid >> 3) + 32, 16 / NR_BITS * (tid & 7));
#elif NR_RECEIVERS_PER_BLOCK == 64
  FetchData<int4> tmpY1((tid >> 2) + 32, 32 / NR_BITS * (tid & 3));
#endif

  tmpY0.load(samples, channel, 0, firstReceiverY, skipLoadYcheck);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
  tmpY1.load(samples, channel, 0, firstReceiverY, skipLoadYcheck);
#endif
  tmpX0.load(samples, channel, 0, firstReceiverX, skipLoadXcheck);
#if NR_RECEIVERS_PER_BLOCK == 48
  tmpX1.load(samples, channel, 0, firstReceiverX, skipLoadXcheck);
#endif

  for (unsigned majorTime = 0; majorTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK; majorTime ++) {
    unsigned buffer = majorTime % NR_SHARED_BUFFERS;

    tmpY0.storeA(aSamples[buffer]);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
    tmpY1.storeA(aSamples[buffer]);
#endif
    tmpX0.storeB(bSamples[buffer]);
#if NR_RECEIVERS_PER_BLOCK == 48
    tmpX1.storeB(bSamples[buffer]);
#endif

    unsigned majorReadTime = majorTime + READ_AHEAD;

    if (majorReadTime < NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK) {
      tmpY0.load(samples, channel, majorReadTime, firstReceiverY, skipLoadYcheck);
#if NR_RECEIVERS_PER_BLOCK == 48 || NR_RECEIVERS_PER_BLOCK == 64
      tmpY1.load(samples, channel, majorReadTime, firstReceiverY, skipLoadYcheck);
#endif
      tmpX0.load(samples, channel, majorReadTime, firstReceiverX, skipLoadXcheck);
#if NR_RECEIVERS_PER_BLOCK == 48
      tmpX1.load(samples, channel, majorReadTime, firstReceiverX, skipLoadXcheck);
#endif
    }

    __syncthreads();

#pragma unroll
    for (unsigned minorTime = 0; minorTime < NR_TIMES_PER_BLOCK; minorTime += ((NR_BITS) == 4 ? 16 : 8)) {
      Afrag aFrag[nrFragmentsY];
      Bfrag bFrag[nrFragmentsX];

      for (unsigned y = 0; y < nrFragmentsY; y ++)
	load_matrix_sync(aFrag[y], &aSamples[buffer][recvYoffset + NR_RECEIVERS_PER_TCM_Y * y][0][minorTime][0], sizeof(aSamples[0][0][0]) * 8 / NR_BITS);

      for (unsigned x = 0; x < nrFragmentsX; x ++)
	load_matrix_sync(bFrag[x], &bSamples[buffer][recvXoffset + NR_RECEIVERS_PER_TCM_X * x][0][0][minorTime][0], sizeof(bSamples[0][0][0][0]) * 8 / NR_BITS);

      for (unsigned y = 0; y < nrFragmentsY; y ++)
	for (unsigned x = 0; x < nrFragmentsX; x ++)
	  mma_sync(sum[y][x], aFrag[y], bFrag[x], sum[y][x]);
    }
  }

#if 0
  for (unsigned y = 0; y < nrFragmentsY; y ++)
    for (unsigned x = 0; x < nrFragmentsX; x ++)
      for (unsigned i = 0; i < sum[0][0].num_storage_elements; i ++)
	if (sum[y][x].x[i] != 0)
#if NR_BITS == 4 || NR_BITS == 8
	  printf("blockIdx=(%d,%d,%d) tid=%u y=%u x=%u i=%u v=%d\n", blockIdx.x, blockIdx.y, blockIdx.z, tid, y, x, i, sum[y][x].x[i]);
#else
	  printf("blockIdx=(%d,%d,%d) tid=%u y=%u x=%u i=%u v=%f\n", blockIdx.x, blockIdx.y, blockIdx.z, tid, y, x, i, sum[y][x].x[i]);
#endif
#endif

#if defined PORTABLE
  __syncthreads();
#endif

  for (unsigned y = 0; y < nrFragmentsY; y ++)
    for (unsigned x = 0; x < nrFragmentsX; x ++)
      storeVisibilities(visibilities, channel, firstReceiverY, firstReceiverX, recvYoffset, recvXoffset, y, x, skipStoreYcheck, skipStoreXcheck, sum[y][x], scratchSpace, tid / warpSize);
}


extern "C" __global__
__launch_bounds__(NR_WARPS * 32, NR_RECEIVERS_PER_BLOCK == 32 ? 4 : 2)
void correlate(Visibilities *visibilities, const Samples *samples, unsigned batchOffset)
{
  const unsigned nrFragmentsY = NR_RECEIVERS_PER_BLOCK / NR_RECEIVERS_PER_TCM_Y / 2;

  unsigned batch = batchOffset + blockIdx.z;
  visibilities += batch;
  samples += batch;
  unsigned block = blockIdx.x;

#if NR_RECEIVERS_PER_BLOCK == 32 || NR_RECEIVERS_PER_BLOCK == 48
  unsigned blockY = (unsigned) (sqrtf(8 * block + 1) - .99999f) / 2;
  unsigned blockX = block - blockY * (blockY + 1) / 2;
  unsigned firstReceiverX = blockX * NR_RECEIVERS_PER_BLOCK;
#elif NR_RECEIVERS_PER_BLOCK == 64
  unsigned blockY = (unsigned) sqrtf(block);
  unsigned blockX = block - blockY * blockY;
  unsigned firstReceiverX = blockX * (NR_RECEIVERS_PER_BLOCK / 2);
#endif
  unsigned firstReceiverY = blockY * NR_RECEIVERS_PER_BLOCK;

  union shared {
    struct {
      alignas(32) SharedData<>::Asamples aSamples;
      alignas(32) SharedData<NR_RECEIVERS_PER_BLOCK == 64 ? 32 : NR_RECEIVERS_PER_BLOCK>::Bsamples bSamples;
    } rectangle;
    struct {
      alignas(32) SharedData<>::Bsamples samples;
    } triangle;
    ScratchSpace scratchSpace[NR_WARPS];
  };

  // the following hack is necessary to run the correlator in the OpenCL environment,
  // as the maximum local memory size is 48K - 16 bytes.  Due to padding in bSamples,
  // the last 16 bytes are not used, so allocate 16 fewer bytes.
  __shared__ char rawbuffer[sizeof(union shared) - 16] __attribute__((aligned(16)));
  union shared &u = (union shared &) rawbuffer;

  if (firstReceiverX == firstReceiverY)
#if NR_RECEIVERS_PER_BLOCK == 32 || NR_RECEIVERS_PER_BLOCK == 48
    doCorrelateRectangle<nrFragmentsY, NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0, NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0, NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK == 0, false>(*visibilities, *samples, firstReceiverY, firstReceiverX, u.rectangle.aSamples, u.rectangle.bSamples, u.scratchSpace);
#elif NR_RECEIVERS_PER_BLOCK == 64
    if (NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK != 0 && (NR_RECEIVERS < NR_RECEIVERS_PER_BLOCK || firstReceiverX >= NR_RECEIVERS / NR_RECEIVERS_PER_BLOCK * NR_RECEIVERS_PER_BLOCK))
      doCorrelateTriangle<false>(*visibilities, *samples, firstReceiverX, 2 * threadIdx.z + threadIdx.y, 64 * threadIdx.z + 32 * threadIdx.y + threadIdx.x, u.triangle.samples, u.scratchSpace);
    else
      doCorrelateTriangle<true>(*visibilities, *samples, firstReceiverX, 2 * threadIdx.z + threadIdx.y, 64 * threadIdx.z + 32 * threadIdx.y + threadIdx.x, u.triangle.samples, u.scratchSpace);
#endif
#if NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK != 0
  else if (NR_RECEIVERS < NR_RECEIVERS_PER_BLOCK || firstReceiverY >= NR_RECEIVERS / NR_RECEIVERS_PER_BLOCK * NR_RECEIVERS_PER_BLOCK)
    doCorrelateRectangle<(NR_RECEIVERS % NR_RECEIVERS_PER_BLOCK + 2 * NR_RECEIVERS_PER_TCM_Y - 1) / NR_RECEIVERS_PER_TCM_Y / 2, false, true, NR_RECEIVERS % (2 * NR_RECEIVERS_PER_TCM_Y) == 0, true>(*visibilities, *samples, firstReceiverY, firstReceiverX, u.rectangle.aSamples, u.rectangle.bSamples, u.scratchSpace);
#endif
  else
    doCorrelateRectangle<nrFragmentsY, true, true, true, true>(*visibilities, *samples, firstReceiverY, firstReceiverX, u.rectangle.aSamples, u.rectangle.bSamples, u.scratchSpace);
}

// TODO: generalise to other values of NR_BITS
extern "C" __global__
__launch_bounds__(NR_WARPS * 32)
void reduce(int2 * __restrict__ out, const long2 * __restrict__ in, unsigned batches)
{
  const unsigned int stride = NR_CHANNELS *NR_BASELINES * NR_POLARIZATIONS * NR_POLARIZATIONS;
  const unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= stride)
    return;
  long2 sum = make_long2(0, 0);
  for (unsigned i = 0; i < batches; i++) {
    long2 value = in[i * stride + idx];
    sum.x += value.x;
    sum.y += value.y;
  }
  // Apply saturation
  sum.x = llmin(llmax(sum.x, -INT_MAX), INT_MAX);
  sum.y = llmin(llmax(sum.y, -INT_MAX), INT_MAX);
  out[idx] = make_complex((int) sum.x, (int) sum.y);
}

} // extern "C++"
