/*******************************************************************************
 * Copyright (c) 2024, National Research Foundation (SARAO)
 *
 * Licensed under the BSD 3-Clause License (the "License"); you may not use
 * this file except in compliance with the License. You may obtain a copy
 * of the License at
 *
 *   https://opensource.org/licenses/BSD-3-Clause
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ******************************************************************************/

extern "C++"  // PyCUDA wraps the whole file in extern "C"
{

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
}

/// Get sizeof and alignof curandStateXORWOW_t
__global__ void sizeof_alignof_curandStateXORWOW_t(int *out)
{
    out[0] = sizeof(hiprandStateXORWOW_t);
    out[1] = alignof(hiprandStateXORWOW_t);
}

/// Initialise an array of curandState_t with sequential sequence numbers
__global__ void init_curandStateXORWOW_t(
    hiprandStateXORWOW_t *out,
    unsigned long long seed,
    unsigned long long sequence_first,
    unsigned long long sequence_step,
    unsigned long long offset,
    unsigned int n)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n)
        return;
    hiprand_init(seed, sequence_first + id * sequence_step, offset, out + id);
}
